#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                   \
  do {                  \
    hipError_t err = stmt;         \
    if (err != hipSuccess) {             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);     \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err)); \
      return -1;              \
    }                   \
  } while (0)

void stencil_cpu(char *_out, char *_in, int width, int height, int depth) {

#define out(i, j, k) _out[((i)*width + (j)) * depth + (k)]
#define in(i, j, k) _in[((i)*width + (j)) * depth + (k)]

  for (int i = 1; i < height - 1; ++i) {
    for (int j = 1; j < width - 1; ++j) {
      for (int k = 1; k < depth - 1; ++k) {
        out(i, j, k) = in(i, j, k + 1) + in(i, j, k - 1) +
          in(i, j + 1, k) + in(i, j - 1, k) +
          in(i + 1, j, k) + in(i - 1, j, k) - 6 * in(i, j, k);
      }
    }
  }
#undef out
#undef in
}

//#define idx3d(i, j, k) (k*height + j) * width + i
#define idx3d(i,j,k) ((i)*width + (j)) * depth + (k)
#define tile_size 16


__device__ float clamp(float val) {
  return (val > 255) ? 255 : (val < 0) ? 0 : val;
}

__global__ void stencil(float *output, float *input, int width, int height,
      int depth) {
  //@@ INSERT CODE HERE
  unsigned int i = blockIdx.z*blockDim.z + threadIdx.z;
  unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;



  __shared__ float zy_2d_cache[tile_size][tile_size];


  float prev = input[idx3d(i,j,0)];
  float current = input[idx3d(i,j,1)];
  float next = input[idx3d(i,j,2)];


  __syncthreads();
  zy_2d_cache[i][j] = current;
  __syncthreads();



  for(auto k = 1u; k < depth-1; k++) {
    auto temp = prev + next +
      (threadIdx.z > 0) ? zy_2d_cache[threadIdx.z - 1][threadIdx.y] : input[idx3d(i-1,j,k)] +
      (threadIdx.z < blockDim.z) ? zy_2d_cache[threadIdx.z + 1][threadIdx.y] : input[idx3d(i+1,j,k)] +
      (threadIdx.y > 0) ? zy_2d_cache[threadIdx.z][threadIdx.y - 1] : input[idx3d(i,j,k-1)] +
      (threadIdx.y < blockDim.y) ? zy_2d_cache[threadIdx.z][threadIdx.y+1] : input[idx3d(i,j,k+1)] -
      6*current;

    output[idx3d(i,j,k)] = clamp(temp);

    prev = current;
    current = next;

    __syncthreads();
    zy_2d_cache[threadIdx.z][threadIdx.y]=next;
    __syncthreads();

    next = input[idx3d(i,j+1,k)];
  }

}

static void launch_stencil(float *deviceOutputData, float *deviceInputData,
         int width, int height, int depth) {
  //@@ INSERT CODE HERE
  //auto len = width*height*depth;
  //auto nThreads = tile_size*tile_size;
  //auto nBlocks = (len+nThreads-1)/nThreads;
  //stencil<<<nBlocks,nThreads>>>(deviceOutputData,deviceInputData,width,height,depth);

  dim3 grid(1,(width-1)/tile_size + 1,(height-1)/tile_size +1);
  dim3 block(1,tile_size,tile_size);

  stencil<<<grid,block>>>(deviceOutputData,deviceInputData,width,height,depth);
}

int main(int argc, char *argv[]) {
  wbArg_t arg;
  int width;
  int height;
  int depth;
  char *inputFile;
  wbImage_t input;
  wbImage_t output;
  float *hostInputData;
  float *hostOutputData;
  float *deviceInputData;
  float *deviceOutputData;

  arg = wbArg_read(argc, argv);

  inputFile = wbArg_getInputFile(arg, 0);

  input = wbImport(inputFile);

  width = wbImage_getWidth(input);
  height = wbImage_getHeight(input);
  depth = wbImage_getChannels(input);

  output = wbImage_new(width, height, depth);
  wbLog(TRACE, "Image size: ", width, " x ", height, " x ", depth);

  hostInputData = wbImage_getData(input);
  hostOutputData = wbImage_getData(output);

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputData,
       width * height * depth * sizeof(float));
  hipMalloc((void **)&deviceOutputData,
       width * height * depth * sizeof(float));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputData, hostInputData,
       width * height * depth * sizeof(float),
       hipMemcpyHostToDevice);
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  launch_stencil(deviceOutputData, deviceInputData, width, height, depth);
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputData, deviceOutputData,
       width * height * depth * sizeof(float),
       hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbSolution(arg, output);

  hipFree(deviceInputData);
  hipFree(deviceOutputData);

  wbImage_delete(output);
  wbImage_delete(input);

  return 0;
}
